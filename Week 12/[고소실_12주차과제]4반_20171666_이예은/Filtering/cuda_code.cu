#include "hip/hip_runtime.h"
#pragma once

#include "cuda_code.cuh"

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL(hipEventDestroy(cuda_timer_start));
	CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

#define Window 2
#define BLOCK_SIZE 4

__constant__ float constant_gaussian_kernel[ 25 ];

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ������� �ʴ´�
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void Gaussian_kernel_no_shared(IN unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int w = floor((double)5 / 2);
	double mean = 0;

	for (int l = -w; l <= w; l++) {
		for (int k = -w; k <= w; k++) {

			if (row + l < 0 || row + l >= height || col + k < 0 || col + k >= width) continue;
			else
				mean += d_bitmaps[(row + l) * width + col +k] * constant_gaussian_kernel[(l + w) * 5 + k + w];		
		}
	}
	d_Gaussian[row*width + col] = mean;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Gaussian ���͸��� �ϴ� Ŀ��
//	shared memory�� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern __shared__ unsigned char sharedBuffer[];
__global__ void Gaussian_kernel_shared(INOUT unsigned char *d_bitmaps, OUT unsigned char *d_Gaussian, long width, long height) {
	/*Todo*/
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;

	const unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
	const unsigned thread_id = threadIdx.y * blockDim.x + threadIdx.x;
	const unsigned id = block_id * BLOCK_SIZE + thread_id;

	int w = 2;// floor((double)5 / 2);
	double mean = 0;
	
	if (thread_id == 0) { // ����
		for (int i = 1; i <= w; i++) {
			if (id - i < 0) {
				for (int j = 0; j < width; j++) {
					sharedBuffer[(w - i) * width + j] = 0;
				}
			}
			else if (id - i >= 0 && id - i <= height) {
				for (int j = 0; j < width; j++)
					sharedBuffer[(w - i) * width + j] = d_bitmaps[(id - i) * width + j];
			}
		}

	}
	/*else if (thread_id % BLOCK_SIZE == 0) { // ���� ��

	}
	else if (thread_id % BLOCK_SIZE == BLOCK_SIZE - 1) { // ������ ��

	}*/
	if (thread_id == BLOCK_SIZE - 1) { // �Ʒ� ��
		for (int i = 1; i <= w; i++) {

			if (id +i >= height) {
				for (int j = 0; j < width; j++)
					sharedBuffer[(BLOCK_SIZE + w + i - 1) * width + j] = 0;
			}
			else if (id + i < height) {
				for (int j = 0; j < width; j++)
					sharedBuffer[(BLOCK_SIZE + w + i - 1) * width + j] = d_bitmaps[(id + i) * width + j];
			}
		}
	}

	for (int i = 0; i < width; i++)
		//sharedBuffer[(thread_id + w) * width + i] = 0; d_bitmaps[id * width + i];
	//sharedBuffer[(thread_id + w) * width] = d_bitmaps[id * width];

	__syncthreads();

	for (int i = 0; i < width; i++) {
		mean = 0;

		for (int l = -w; l <= w; l++) {
			for (int k = -w; k <= w; k++) {

				if (i + l >= width || i + l < 0) continue;
				//else
					//mean += sharedBuffer[(thread_id + w + k) * width + i + l] * constant_gaussian_kernel[(k + w) * 5 + l + w]; // error
				//printf("%lf\n", constant_gaussian_kernel[(l + w) * 5 + k + w]);
			}
		}
		//d_Gaussian[id * width + i] = mean;
	}
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Constant variable �� gaussian kernel�� �����ϴ� �Լ�
//	�Ŀ� gaussian filtering ���� ����Ѵ�.
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void Set_Gaussain_Kernel(){
	float _1 = 1.0f / 256.0f;
	float _4 = _1 * 4;
	float _6 = _1 * 6;
	float _16 = _1 * 16;
	float _24 = _1 * 24;
	float _36 = _1 * 36;

	float *p_gaussian_kernel = new float[25];

	p_gaussian_kernel[0] = p_gaussian_kernel[4] = p_gaussian_kernel[20] = p_gaussian_kernel[24] = _1;
	p_gaussian_kernel[1] = p_gaussian_kernel[3] = p_gaussian_kernel[5] = p_gaussian_kernel[9]= _4;
	p_gaussian_kernel[15] = p_gaussian_kernel[19] = p_gaussian_kernel[21] = p_gaussian_kernel[23] = _4;
	p_gaussian_kernel[2] = p_gaussian_kernel[10] = p_gaussian_kernel[14] = p_gaussian_kernel[22] = _6;
	p_gaussian_kernel[6] = p_gaussian_kernel[8] = p_gaussian_kernel[16] = p_gaussian_kernel[18] = _16;
	p_gaussian_kernel[7] = p_gaussian_kernel[11] =p_gaussian_kernel[13] = p_gaussian_kernel[17] = _24;
	p_gaussian_kernel[12] = _36;

	hipMemcpyToSymbol(HIP_SYMBOL( constant_gaussian_kernel), p_gaussian_kernel, sizeof( float ) * 25 );

	delete[] p_gaussian_kernel;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//	
//	Ŀ���� �����ϱ� �� �ʿ��� �ڷ�� �غ� �� Ŀ���� ������ ����̽��� ����
//	Shared_flag �Է� �� NO_SHARED �� SHARED �� �� ���� ��ũ�θ� ������
//	flag���� �´� Ŀ���� ����
//	
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

float Do_Gaussian_on_GPU(IN unsigned char *p_bitmaps, OUT unsigned char *p_Gaussian, long width, long height, int Shared_flag)
{
	/*Todo*/
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	CUDA_CALL(cudaStatus);

	Set_Gaussain_Kernel();
	unsigned char* d_bitmaps, * d_Gaussian;
	size_t mem_size;

	mem_size = width * height * sizeof(unsigned char);
	CUDA_CALL(hipMalloc(&d_bitmaps, mem_size));
	CUDA_CALL(hipMemcpy(d_bitmaps, p_bitmaps, mem_size, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_Gaussian, mem_size));

	dim3 blockDIm(BLOCK_SIZE, BLOCK_SIZE);
	dim3 gridDim(width / blockDIm.x, height / blockDIm.y);
	CHECK_TIME_INIT_GPU();
	CHECK_TIME_START_GPU();

	switch (Shared_flag)
	{
	case NO_SHARED:
		Gaussian_kernel_no_shared << <gridDim, blockDIm >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	case SHARED:
		Gaussian_kernel_shared << <gridDim, blockDIm, sizeof(unsigned char)* (BLOCK_SIZE * BLOCK_SIZE + (BLOCK_SIZE + 4) * width) >> > (d_bitmaps, d_Gaussian, width, height);
		break;
	}

	CUDA_CALL(cudaStatus = hipDeviceSynchronize());
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU();

	CUDA_CALL(hipMemcpy(p_Gaussian, d_Gaussian, mem_size, hipMemcpyDeviceToHost));

	hipFree(d_bitmaps);
	hipFree(d_Gaussian);

	return device_time;
}