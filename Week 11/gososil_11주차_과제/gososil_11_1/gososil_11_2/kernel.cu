#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 128
#define N_EQUATIONS (1 << 26)
#define EPSILON 1

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float* A;
float* B;
float* C;
float* X0_CPU; float* X1_CPU;
float* FX0_CPU; float* FX1_CPU;
float* X0_GPU; float* X1_GPU;
float* FX0_GPU; float* FX1_GPU;

hipError_t find_roots_GPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n);

__global__ void Find_Root_Kernel(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n)
{
    /*Thread Idx*/
    //int row = blockDim.y * blockIdx.y + threadIdx.y;
    //int col = blockDim.x * blockIdx.x + threadIdx.x;
    //int tid = gridDim.x * blockDim.x * row + col;
    int tid = blockIdx.x* blockDim.x + threadIdx.x;

    float a, b, c, d, x0, x1, tmp;
    a = A[tid]; b = B[tid]; c = C[tid];
    d = sqrtf(b * b - 4.0f * a * c);
    tmp = 1.0f / (2.0f * a);
    X0[tid] = x0 = (-b - d) * tmp;
    X1[tid] = x1 = (-b + d) * tmp;
    FX0[tid] = (a * x0 + b) * x0 + c;
    FX1[tid] = (a * x1 + b) * x1 + c;
}


void find_roots_CPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n) {

    int i;
    float a, b, c, d, x0, x1, tmp;
    for (i = 0; i < n; i++) {
        a = A[i]; b = B[i]; c = C[i];
        d = sqrtf(b * b - 4.0f * a * c);
        tmp = 1.0f / (2.0f * a);
        X0[i] = x0 = (-b - d) * tmp;
        X1[i] = x1 = (-b + d) * tmp;
        FX0[i] = (a * x0 + b) * x0 + c;
        FX1[i] = (a * x1 + b) * x1 + c;
    }  
}

void init_bin_file(int size) {
    srand((unsigned)time(NULL));
    printf("***Binary File init Start!!\n");
    FILE *fpa = fopen("a.bin", "wb");
    FILE* fpb = fopen("b.bin", "wb");
    FILE* fpc = fopen("c.bin", "wb");

    fwrite(&size, sizeof(int), 1, fpa);
    fwrite(&size, sizeof(int), 1, fpb);
    fwrite(&size, sizeof(int), 1, fpc);

    float random_a; float random_b; float random_c;
    for (int i = 0; i < size; i++) {
        random_a = (float)(((double)rand() / RAND_MAX) * 40.0f - 20.0f);
        random_b = (float)(((double)rand() / RAND_MAX) * 40.0f - 20.0f);
        random_c = (float)(((double)rand() / RAND_MAX) *40.0f - 20.0f);

        if (random_b * random_b < random_a * random_c * 4.0f) {
            i--;
            continue;
        }

        fwrite(&random_a, sizeof(float), 1, fpa);
        fwrite(&random_b, sizeof(float), 1, fpb);
        fwrite(&random_c, sizeof(float), 1, fpc);
    }

    fclose(fpa);
    fclose(fpb);
    fclose(fpc);
    printf("***Binary File init End!!\n\n");
}

void read_bin_file()
{
    printf("***Binary File Read Start!!\n");
    FILE* fpa = fopen("a.bin", "rb");
    FILE* fpb = fopen("b.bin", "rb");
    FILE* fpc = fopen("c.bin", "rb");
    /*Todo*/
    fread(&N, sizeof(int), 1, fpa);
    fread(&N, sizeof(int), 1, fpb);
    fread(&N, sizeof(int), 1, fpc);

    A = new float[N];
    B = new float[N];
    C = new float[N];
    X0_CPU = new float[N]; X1_CPU = new float[N];
    FX0_CPU = new float[N]; FX1_CPU = new float[N];
    X0_GPU = new float[N]; X1_GPU = new float[N];
    FX0_GPU = new float[N]; FX1_GPU = new float[N];

    fread(A, sizeof(float), N, fpa);
    fread(B, sizeof(float), N, fpb);
    fread(C, sizeof(float), N, fpc);

    fclose(fpa);
    fclose(fpb);
    fclose(fpc);
    printf("***Binary File Read End!!\n\n");
}

int main()
{
    init_bin_file(N_EQUATIONS);
    read_bin_file();

    printf("The problem size is %d.\n", N);
    int i;

    //CPU 
    printf("***find_roots_CPU Start!!\n");
    CHECK_TIME_START;
    find_roots_CPU(A, B, C, X0_CPU, X1_CPU, FX0_CPU, FX1_CPU, N);
    CHECK_TIME_END(compute_time);
    printf("***find_roots_CPU End!!\n\n");

    printf("CPU time = %.6f\n", compute_time);
    //GPU

    printf("***find_roots_GPU Start!!\n");
    find_roots_GPU(A, B, C, X0_GPU, X1_GPU, FX0_GPU, FX1_GPU, N);
    printf("***find_roots_GPU End!!\n\n");

    printf("GPU time = %.6f\n", device_time);

    for (i = 0; i < N; i++)
    {
        if (fabs(FX0_CPU[i]) > EPSILON || fabs(FX1_CPU[i]) > EPSILON || fabs(FX0_GPU[i]) > EPSILON || fabs(FX1_GPU[i]) > EPSILON)
        {
            printf("%d\n", i);
            printf("%f != %f || %f != %f\n", FX0_CPU[i], FX1_CPU[i], FX0_GPU[i], FX1_GPU[i]);
            break;
        }
    }

    if (i == N)
        printf("***Kernel execution Success!!\n\n");

    // Write the output array into the output file.
    FILE* fpx0 = fopen("X0.bin", "wb");
    FILE* fpx1 = fopen("X1.bin", "wb");
    FILE* fpfx0 = fopen("FX0.bin", "wb");
    FILE* fpfx1 = fopen("FX1.bin", "wb");

    if (!fpx0 || !fpx1 || !fpfx0 || !fpfx1) {
        fprintf(stderr, "Error: cannot open the output file...\n");
        exit(-1);
    }

    fwrite(&N, sizeof(int), 1, fpx0);
    fwrite(&N, sizeof(int), 1, fpx1);
    fwrite(&N, sizeof(int), 1, fpfx0);
    fwrite(&N, sizeof(int), 1, fpfx1);

    for (i = 0; i < N; i++) {
        fwrite(&X0_GPU[i], sizeof(float), 1, fpx0);
        fwrite(&X1_GPU[i], sizeof(float), 1, fpx1);
        fwrite(&FX0_GPU[i], sizeof(float), 1, fpfx0);
        fwrite(&FX1_GPU[i], sizeof(float), 1, fpfx1);
    }

    fclose(fpx0);
    fclose(fpx1);
    fclose(fpfx0);
    fclose(fpfx1);
    printf("end!!\n\n");

    return 0;
}


hipError_t find_roots_GPU(float* A, float* B, float* C,
    float* X0, float* X1, float* FX0, float* FX1, int n)
{
    float* copy_a;
    float* copy_b;
    float* copy_c;
    float* copy_x0;
    float* copy_x1;
    float* copy_fx0;
    float* copy_fx1;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    size_t size = n * sizeof(float);
    CUDA_CALL(hipMalloc(&copy_a, size));
    CUDA_CALL(hipMemcpy(copy_a, A, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&copy_b, size));
    CUDA_CALL(hipMemcpy(copy_b, B, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMalloc(&copy_c, size));
    CUDA_CALL(hipMemcpy(copy_c, C, size, hipMemcpyHostToDevice));

    CUDA_CALL(hipMalloc(&copy_x0, size));
    CUDA_CALL(hipMalloc(&copy_x1, size));
    CUDA_CALL(hipMalloc(&copy_fx0, size));
    CUDA_CALL(hipMalloc(&copy_fx1, size));

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(N / dimBlock.x);

    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    Find_Root_Kernel << < dimGrid, dimBlock >> > (copy_a, copy_b, copy_c, copy_x0, copy_x1, copy_fx0, copy_fx1, N);

    CHECK_TIME_END_GPU(device_time)
    CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(X0, copy_x0, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(X1, copy_x1, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(FX0, copy_fx0, size, hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(FX1, copy_fx1, size, hipMemcpyDeviceToHost));

        Error:
    hipFree(copy_a);
    hipFree(copy_b);
    hipFree(copy_c);
    hipFree(copy_x0);
    hipFree(copy_x1);
    hipFree(copy_fx0);
    hipFree(copy_fx1);

    return cudaStatus;
}